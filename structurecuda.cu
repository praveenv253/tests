#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

struct Test {
	int x;
	int y;
};

__global__ void testfunc(Test t, int *z)
{
	*z = t.x + t.y;
}

int main()
{
	Test t;
	t.x = 10;
	t.y = 20;
	
	int *z;
	hipMalloc((void **)&z, sizeof(int));
	testfunc<<<1, 1>>>(t, z);

	int hz;
	hipMemcpy(&hz, z, sizeof(int), hipMemcpyDeviceToHost);
	cout<<hz<<endl;

	return 0;
}
